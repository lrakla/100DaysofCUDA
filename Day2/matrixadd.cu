#include "hip/hip_runtime.h"
//%%writefile matrixadd.cu
#include <iostream>

__global__ void matAddKernel(float *A, float *B, float *C, int N)
{
    int col = blockDim.x*blockIdx.x + threadIdx.x;
    int row = blockDim.y*blockIdx.y + threadIdx.y;

    if(col < N  && row < N)
    {
        C[row*N + col] = A[row*N + col] + B[row*N + col];
    }
}


int main()
{
    const int N = 1<<5;
    float *A, *B, *C;

    A = (float *)malloc(N*N*sizeof(float));
    B = (float *)malloc(N*N*sizeof(float));
    C = (float *)malloc(N*N*sizeof(float));

    for(int i = 0; i < N; i++)
    {
        for(int j =0; j< N; j++)
        {
            A[i*N + j] = 1.0f;
            B[i*N + j] = 2.0f;
            C[i*N + j] = 0.0f;
        }
    }
    
    float *A_d, *B_d, *C_d;
    hipMalloc((void**)&A_d, N*N*sizeof(float));
    hipMalloc((void**)&B_d, N*N*sizeof(float));
    hipMalloc((void**)&C_d, N*N*sizeof(float));

    hipMemcpy(A_d, A, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, N*N*sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(16,16);
    dim3 gridDim(ceil(N/16.0f), ceil(N/16.0f)); // Be careful about integer division here
    
    matAddKernel<<<gridDim,blockDim>>>(A_d,B_d,C_d,N);
    hipDeviceSynchronize();

    hipMemcpy(C, C_d, N*N*sizeof(float), hipMemcpyDeviceToHost);

    // verify C is 3.0
    for(int i = 0; i < N; i++)
    {
        for(int j = 0; j < N; j++)
        {
            if(C[i*N + j] != 3.0f)
            {
                std::cout << "Error: mismatch at position " << i << " " << j << std 
                << std::endl;
                break;
            }
        }
    }

    std::cout << "Matrix addition completed successfully!" << std::endl;

    free(A);
    free(B);
    free(C);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

}